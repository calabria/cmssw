#include "hip/hip_runtime.h"
// CUDA-specific test resources
#include <iostream>
#include "FWCore/Services/interface/utils/cuda_launch_configuration.cuh"

#define BLOCK_SIZE 32

__global__ void long_kernel(const int n, const int times, const float* in, float* out)
{
  int x= blockIdx.x*blockDim.x + threadIdx.x;
  if (x < n){
    out[x]= 0;
    for(int i=0; i<times; i++){
      out[x]+= in[x];
    }
  }
}
__global__ void matAdd_kernel(int m, int n, const float* __restrict__ A, 
                              const float* __restrict__ B, float* __restrict__ C)
{
  int x= blockIdx.x*blockDim.x + threadIdx.x;
  int y= blockIdx.y*blockDim.y + threadIdx.y;

  // ### Difference between manual and automatic kernel grid:
  if (x<n && y<m)
    C[y*n+x]= A[y*n+x]+B[y*n+x];
  //if (y*n+x < n*m)
    //C[y*n+x]= A[y*n+x]+B[y*n+x];
}
__global__ void original_kernel(unsigned meanExp, float* cls, float* clx, float* cly)
{
  int i= blockDim.x*blockIdx.x+threadIdx.x;
  if(i<meanExp){
    if (cls[i] != 0){
      clx[i] /= cls[i];
      cly[i] /= cls[i];
    }
    cls[i]= 0;
  }
}
__global__
void simpleTask_GPU(unsigned meanExp, float* cls, float* clx, float* cly)
{
  unsigned i= blockDim.x*blockIdx.x+threadIdx.x;
  if(i<meanExp){
    if (cls[i] != 0){
      clx[i] /= cls[i];
      cly[i] /= cls[i];
    }
    cls[i]= 0;
  }
}
void simpleTask_CPU(unsigned meanExp, float* cls, float* clx, float* cly)
{
  for (unsigned int subcl_idx = 0;
       subcl_idx < meanExp; subcl_idx++){
    if (cls[subcl_idx] != 0) {
      clx[subcl_idx] /= cls[subcl_idx];
      cly[subcl_idx] /= cls[subcl_idx];
    }
    cls[subcl_idx] = 0;
  }
}

//@@@@@@@@@@@@@@@@
  void long_auto(bool gpu, unsigned& launchSize,
                 const int n, const int times, const float* in, float* out){
    auto execPol= cuda::AutoConfig()(launchSize, (void*)long_kernel);
    long_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(n,times,in,out);
  }
  void matAdd_auto(bool gpu, unsigned& launchSize,
                   int m, int n, const float* __restrict__ A, 
                   const float* __restrict__ B, float* __restrict__ C){
    auto execPol= cuda::AutoConfig()(launchSize, (void*)matAdd_kernel);
    matAdd_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(m,n,A,B,C);
  }
  void original_auto(bool gpu, unsigned& launchSize,
                     unsigned meanExp, float* cls, float* clx, float* cly){
    auto execPol= cuda::AutoConfig()(launchSize, (void*)original_kernel);
    original_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(meanExp,cls,clx,cly);
  }
//@@@@@@@@@@@@@@@@
  void long_man(bool gpu, const cuda::ExecutionPolicy& execPol,
                const int n, const int times, const float* in, float* out){
    long_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(n,times,in,out);
  }
  void matAdd_man(bool gpu, const cuda::ExecutionPolicy& execPol,
                  int m, int n, const float* __restrict__ A,
                  const float* __restrict__ B, float* __restrict__ C){
    matAdd_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(m,n,A,B,C);
  }
  void original_man(bool gpu, const cuda::ExecutionPolicy& execPol,
                    unsigned meanExp, float* cls, float* clx, float* cly){
    original_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(meanExp,cls,clx,cly);
  }
