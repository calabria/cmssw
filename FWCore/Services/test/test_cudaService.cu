#include "hip/hip_runtime.h"
//! Kernels and kernel wrappers used by the CudaService test suite
//! @sa test_cudaService_gcc.cppunit.cc
#include "FWCore/Services/interface/utils/cuda_execution_policy.h"
#include "FWCore/Services/interface/utils/cuda_pointer.h"

#define BLOCK_SIZE 32

//@@@@@@@@@@@@@@@@ KERNELS
__global__ void long_kernel(const int n, const int times, const float* in, float* out)
{
  int x= blockIdx.x*blockDim.x + threadIdx.x;
  if (x < n){
    out[x]= 0;
    for(int i=0; i<times; i++){
      out[x]+= in[x];
    }
  }
}
__global__ void matAdd_kernel(int m, int n, const float* __restrict__ A, 
                              const float* __restrict__ B, float* __restrict__ C)
{
  int x= blockIdx.x*blockDim.x + threadIdx.x;
  int y= blockIdx.y*blockDim.y + threadIdx.y;

  // ### Difference between manual and automatic kernel grid:
  if (x<n && y<m)
    C[y*n+x]= A[y*n+x]+B[y*n+x];
  //if (y*n+x < n*m)
    //C[y*n+x]= A[y*n+x]+B[y*n+x];
}
__global__ void original_kernel(unsigned meanExp, float* cls, float* clx, float* cly)
{
  int i= blockDim.x*blockIdx.x+threadIdx.x;
  if(i<meanExp){
    if (cls[i] != 0){
      clx[i] /= cls[i];
      cly[i] /= cls[i];
    }
    cls[i]= 0;
  }
}

//@@@@@@@@@@@@@@@@ AUTO WRAPPERS (without fallbacks)
  void long_auto(bool gpu, unsigned& launchSize,
                 const int n, const int times, const float* in, float* out){
    auto execPol= cuda::AutoConfig()(launchSize, (void*)long_kernel);
    if(gpu) long_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(n,times,in,out);
  }
  void matAdd_auto(bool gpu, unsigned& launchSize,
                   int m, int n, const float* __restrict__ A, 
                   const float* __restrict__ B, float* __restrict__ C){
    auto execPol= cuda::AutoConfig()(launchSize, (void*)matAdd_kernel);
    if(gpu) matAdd_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(m,n,A,B,C);
  }
  void original_auto(bool gpu, unsigned& launchSize,
                     unsigned meanExp, float* cls, float* clx, float* cly){
    auto execPol= cuda::AutoConfig()(launchSize, (void*)original_kernel);
    if(gpu) original_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(meanExp,cls,clx,cly);
  }
//@@@@@@@@@@@@@@@@ MANUAL WRAPPERS
  void long_man(bool gpu, const cuda::ExecutionPolicy& execPol,
                const int n, const int times, const float* in, float* out){
    if(gpu) long_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(n,times,in,out);
  }
  void matAdd_man(bool gpu, const cuda::ExecutionPolicy& execPol,
                  int m, int n, const float* __restrict__ A,
                  const float* __restrict__ B, float* __restrict__ C){
    if(gpu) matAdd_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(m,n,A,B,C);
  }
  void original_man(bool gpu, const cuda::ExecutionPolicy& execPol,
                    unsigned meanExp, float* cls, float* clx, float* cly){
    if(gpu) original_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(meanExp,cls,clx,cly);
  }

//@@@@@@@@@@@@@@@@ FALLBACK for "original_kernel"
void original_CPU(unsigned meanExp, float* cls, float* clx, float* cly)
{
  for (unsigned int subcl_idx = 0;
       subcl_idx < meanExp; subcl_idx++){
    if (cls[subcl_idx] != 0) {
      clx[subcl_idx] /= cls[subcl_idx];
      cly[subcl_idx] /= cls[subcl_idx];
    }
    cls[subcl_idx] = 0;
  }
}

//@@@@@@@@@@@@@@@@ FALLBACK for "original_kernel"
struct KernelData{
  int a, b;
  cudaPointer<float[]> arrayIn;
  cudaPointer<float[]> arrayOut;
};
// "cudaPointer[]: calling host function from kernel"
// "illegal memory access encountered"?
__global__ void actOnStructKernel(KernelData* data){
  int i= blockDim.x*blockIdx.x+threadIdx.x;
  if (i < data->arrayIn.size(true))
    data->arrayOut.at(i)= data->arrayIn.at(i)+data->a*data->b;
}
void actOnStructWrapper(bool gpu, const cuda::ExecutionPolicy& execPol,
                        KernelData* data){
  if(gpu) actOnStructKernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(data);
}

