#include "hip/hip_runtime.h"
// CUDA-specific test resources
#include <iostream>
#include "FWCore/Services/interface/utils/cuda_launch_configuration.cuh"

#define BLOCK_SIZE 32

__global__ void longKernel(const int n, const int times, const float* in, float* out)
{
  int x= blockIdx.x*blockDim.x + threadIdx.x;
  if (x < n){
    out[x]= 0;
    for(int i=0; i<times; i++){
      out[x]+= in[x];
    }
  }
}
__global__ void matAddKernel(int m, int n, const float* __restrict__ A, 
                              const float* __restrict__ B, float* __restrict__ C)
{
  int x= blockIdx.x*blockDim.x + threadIdx.x;
  int y= blockIdx.y*blockDim.y + threadIdx.y;

  // ### Difference between manual and automatic kernel grid:
  if (x<n && y<m)
    C[y*n+x]= A[y*n+x]+B[y*n+x];
  //if (y*n+x < n*m)
    //C[y*n+x]= A[y*n+x]+B[y*n+x];
}
__global__ void originalKernel(unsigned meanExp, float* cls, float* clx, float* cly)
{
  int i= blockDim.x*blockIdx.x+threadIdx.x;
  if(i<meanExp){
    if (cls[i] != 0){
      clx[i] /= cls[i];
      cly[i] /= cls[i];
    }
    cls[i]= 0;
  }
}

void cudaTaskImplement(int n, int i, const float* din, int times){
  float *dout;
  hipMalloc((void **) &dout, n*sizeof(float));
  dim3 grid((n-1)/BLOCK_SIZE/BLOCK_SIZE+1);
  dim3 block(BLOCK_SIZE*BLOCK_SIZE);
  longKernel<<<grid,block>>>(n, times, din, dout);
  hipStreamSynchronize(hipStreamPerThread);
  float out;
  hipMemcpy(&out, dout+i, 1*sizeof(float), hipMemcpyDeviceToHost);
  std::cout << "GPU::" << out << "\t";
  hipFree(dout);
}
