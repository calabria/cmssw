#include "hip/hip_runtime.h"
// Service to test
#include "FWCore/Services/interface/thread_pool_TBBQueueBlocking.h"

// std
#include <iostream>
#include <vector>
#include <future>
#include <mutex>
#include <atomic>
#include <condition_variable>

#include <algorithm>

#include <thread>
#include <chrono>

// CMSSW
#include "FWCore/ServiceRegistry/interface/Service.h"
#include "FWCore/ServiceRegistry/interface/ServiceRegistry.h"

#include "FWCore/PluginManager/interface/standard.h"
#include "FWCore/PluginManager/interface/PluginManager.h"

// cppunit-specific
#include "cppunit/extensions/HelperMacros.h"
#include "Utilities/Testing/interface/CppUnit_testdriver.icpp"

using namespace std;
using namespace edm;

class TestThreadPoolService: public CppUnit::TestFixture {
  CPPUNIT_TEST_SUITE(TestThreadPoolService);
  CPPUNIT_TEST(basicUseTest);
  //CPPUNIT_TEST(passServiceArgTest);
  CPPUNIT_TEST(CUDATest);
  CPPUNIT_TEST(CUDAAutolaunchManagedTest);
  CPPUNIT_TEST(timeBenchmark);
  CPPUNIT_TEST_SUITE_END();
public:
  void setUp();
  void tearDown() {
    //(*poolPtr)->clearTasks();
    cout<<"\n";
  }
  void basicUseTest();
  //!< @brief Test behaviour if the task itself enqueues another task in same pool
  void passServiceArgTest();
  //!< @brief Test scheduling many threads that launch CUDA kernels
  void CUDATest();
  //!< @brief Test auto launch cuda kernel with its arguments in managed memory
  void CUDAAutolaunchManagedTest();
  void timeBenchmark();
private:
  void print_id(int id);
  void go();
  void cudaTask(int n, int i, const float* din, int times);
  //--$--//
  mutex mtx;
  condition_variable cv;
  bool ready= false;
  long sum= 0;
  const int BLOCK_SIZE= 32;

  ServiceToken serviceToken;
  unique_ptr<Service<service::ThreadPoolService>> poolPtr;
  unique_ptr<ServiceRegistry::Operate> operate;
};

///registration of the test so that the runner can find it
CPPUNIT_TEST_SUITE_REGISTRATION(TestThreadPoolService);

__global__ void longKernel(const int n, const int times, const float* in, float* out)
{
  int x= blockIdx.x*blockDim.x + threadIdx.x;
  if (x < n){
    out[x]= 0;
    for(int i=0; i<times; i++){
      out[x]+= in[x];
    }
  }
}
void TestThreadPoolService::setUp(){
  static atomic_flag notFirstTime= ATOMIC_FLAG_INIT;
  if (!notFirstTime.test_and_set()){
    // Init modelled after "FWCore/Catalog/test/FileLocator_t.cpp"
    // Make the services.
    edmplugin::PluginManager::configure(edmplugin::standard::config());
    //serviceToken= edm::ServiceRegistry::createServicesFromConfig(serviceConfig);
    ParameterSet pSet;
    pSet.addParameter("@service_type", string("ThreadPoolService"));
    vector<ParameterSet> vec;
    vec.push_back(pSet);
    operate= unique_ptr<ServiceRegistry::Operate>(
        new ServiceRegistry::Operate(edm::ServiceRegistry::createSet(vec)));
    poolPtr= unique_ptr<Service<service::ThreadPoolService>>(
        new Service<service::ThreadPoolService>);
    //(*poolPtr)->startWorkers();
    cout<<"[ThreadPoolServiceTest::init] Service initialized\n";
  }
}
void TestThreadPoolService::print_id(int id) {
  unique_lock<mutex> lck(mtx);
  while (!ready) cv.wait(lck);
  // ...
  cout << id << "\t";
  sum+= id;
}
void TestThreadPoolService::go() {
  unique_lock<mutex> lck(mtx);
  ready = true;
  cv.notify_all();
}
void TestThreadPoolService::cudaTask(int n, int i, const float* din, int times){
  float *dout;
  hipMalloc((void **) &dout, n*sizeof(float));
  dim3 grid((n-1)/BLOCK_SIZE/BLOCK_SIZE+1);
  dim3 block(BLOCK_SIZE*BLOCK_SIZE);
  longKernel<<<grid,block>>>(n, times, din, dout);
  hipStreamSynchronize(hipStreamPerThread);
  float out;
  hipMemcpy(&out, dout+i, 1*sizeof(float), hipMemcpyDeviceToHost);
  cout << "GPU::" << out << "\t";
  hipFree(dout);
}

void TestThreadPoolService::basicUseTest()
{
  cout<<"\nStarting basic test...\n";
  (*poolPtr)->getFuture([]() {cout<<"Empty task\n";}).get();
  vector<future<void>> futures;
  const int N= 30;

  // spawn N threads:
  for (int i=0; i<N; ++i)
    futures.emplace_back((*poolPtr)->getFuture(&TestThreadPoolService::print_id, this,i+1));
  go();

  for (auto& future: futures) future.get();
  cout << "\n[basicUseTest] DONE, sum= "<<sum<<"\n";
	for(int i=0; i<N; i++)
		sum-= i+1;
  CPPUNIT_ASSERT_EQUAL(sum, 0l);
}
void TestThreadPoolService::passServiceArgTest()
{
  cout<<"\nStarting passServiceArg test...\n"
      <<"(requires >1 thread, otherwise will never finish)\n";
  (*poolPtr)->getFuture([&]() {
    cout<<"Recursive enqueue #1\n";
    //ServiceRegistry::Operate operate(serviceToken);
    (*poolPtr)->getFuture([]() {cout<<"Pool service captured\n";}).get();
  }).get();
  (*poolPtr)->getFuture([this](Service<service::ThreadPoolService> poolArg){
    cout<<"Recursive enqueue #2\n";
    //ServiceRegistry::Operate operate(serviceToken);
    poolArg->getFuture([]() {cout<<"Pool service passed as arg\n";}).get();
  }, (*poolPtr)).get();
}
void TestThreadPoolService::CUDATest()
{
  cout<<"\nStarting CUDA test...\n";
  vector<future<void>> futures;
  const int N= 30;

  float *in, *din;
  int n= 2000;
  in= new float[n];
  for(int i=0; i<n; i++) in[i]= 10*cos(3.141592/100*i);
  // Make GPU input data available for all threads
  hipMalloc((void **) &din, n*sizeof(float));
  hipMemcpy(din, in, n*sizeof(float), hipMemcpyHostToDevice);

  // spawn N threads
  for (int i=0; i<N; ++i){
    futures.emplace_back((*poolPtr)->getFuture(&TestThreadPoolService::cudaTask, this,
                         n, i, din, 2));
  }
  for (auto& future: futures) future.get();
}

#define TOLERANCE 5e-1
void TestThreadPoolService::CUDAAutolaunchManagedTest()
{
  cout<<"\nStarting CUDA autolaunch (managed) test...\n";
  float *in, *out;
  const int n= 10000000, times= 1000;
  hipMallocManaged(&in, n*sizeof(float));  //hipMemAttachHost?
  hipMallocManaged(&out, n*sizeof(float));
  for(int i=0; i<n; i++) in[i]= 10*cos(3.141592/100*i);

  cout<<"Launching auto...\n";
  // Auto launch config
  cudaConfig::ExecutionPolicy execPol((*poolPtr)->configureLaunch(n, longKernel));
  (*poolPtr)->cudaLaunchManaged(execPol, longKernel, (int)n,(int)times,
                          const_cast<const float*>(in),out).get();
  for(int i=0; i<n; i++) if (times*in[i]-out[i]>TOLERANCE || times*in[i]-out[i]<-TOLERANCE){
    cout<<"ERROR: i="<<i<<'\n';
    CPPUNIT_ASSERT_DOUBLES_EQUAL(times*in[i], out[i], TOLERANCE);
  }

  cout<<"Launching manual...\n";
  // Manual launch config
  execPol= cudaConfig::ExecutionPolicy(320, (n-1+320)/320);
  (*poolPtr)->cudaLaunchManaged(execPol, longKernel, (int)n,(int)times,
                          const_cast<const float*>(in),out).get();
  for(int i=0; i<n; i++) if (times*in[i]-out[i]>TOLERANCE || times*in[i]-out[i]<-TOLERANCE){
    cout<<"ERROR: i="<<i<<'\n';
    CPPUNIT_ASSERT_DOUBLES_EQUAL(times*in[i], out[i], TOLERANCE);
  }

  hipFree(in);
  hipFree(out);
}

void TestThreadPoolService::timeBenchmark()
{
  cout << "Starting quick time benchmark...\n";
  long N= 10000000;
  auto start= chrono::steady_clock::now();
  auto end = start;
  auto diff= start-start;
  future<void> fut;
  int threadN= std::thread::hardware_concurrency();

  vector<future<void>> futVec(threadN);
  diff= start-start;
  for (int i = 0; i <= N/threadN; ++i)
  {
    start = chrono::steady_clock::now();
    for(register int thr=0; thr<threadN; thr++)
      futVec[thr]= (*poolPtr)->getFuture([] (){
        this_thread::sleep_for(chrono::microseconds(1));
      });
    for_each(futVec.begin(), futVec.end(), [] (future<void>& elt) {
      elt.get();
    });
    end = chrono::steady_clock::now();

    diff += (i>0)? end-start: start-start;
  }
  cout << "ThreadPoolService normal operation: "<< chrono::duration <double, nano> (diff).count()/N << " ns" << endl;
}
