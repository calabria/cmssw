#include "hip/hip_runtime.h"
#include "FWCore/Services/interface/utils/cuda_launch_configuration.cuh"

// <name>_kernel(...)
__global__ void simpleTask_kernel(unsigned meanExp, float* cls, float* clx, float* cly)
{
  unsigned i= blockDim.x*blockIdx.x+threadIdx.x;
  if(i<meanExp){
    if (cls[i] != 0){
      clx[i] /= cls[i];
      cly[i] /= cls[i];
    }
    cls[i]= 0;
  }
}

//@@@@@@@@@@@@@@@@@@@@@@@@@@

// <name>_auto(launchSize, ...) (1D)
void simpleTask_auto(unsigned& launchSize, unsigned meanExp, float* cls, float* clx, float* cly)
{
  auto execPol= cudaConfig::configure(true, launchSize, simpleTask_kernel);
  simpleTask_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(
                  meanExp, cls, clx, cly);
}
// <name>(execPol, ...)
void simpleTask_man(const cudaConfig::ExecutionPolicy& execPol,
                    unsigned meanExp, float* cls, float* clx, float* cly)
{
  simpleTask_kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(
                    meanExp, cls, clx, cly);
}
// <name>_config(launchSize)
cudaConfig::ExecutionPolicy simpleTask_config(bool cudaStatus, int launchSize){
  return cudaConfig::configure(cudaStatus, launchSize, simpleTask_kernel);
}

//@@@@@@@@@@@@@@@@@@@@@@@@@


// GenerateKernelWrappers(simpleTask) ->
// simpleTask_auto
// simpleTask
// simpleTask_config


////////////////////////////////////////////////////////////////////////////////
//class KernelWrap{};
//  simpleTask_auto(launchSize,...)= 
// template<typename F, typename... Args>
// autoLaunch(size, F&& kernel, Args... args){
//   auto execPol= cudaConfig::configure<???>(true, launchSize, (???)simpleTask_kernel);


//   hipLaunchKernel()
// }


// hipLaunchKernel(reinterpret_cast<const void*>( const void* func), dim3 gridDim, dim3 blockDim, 
//                   void** args, size_t sharedMem, hipStream_t stream )

//@@@@@@@@@@@@@@@@@@@@@@@@@
#define GenerateKernelWrappers(kernelName, args...) \
void kernelName ## _auto(int launchSize, ##args){\
  auto execPol= cudaConfig::configure(true, launchSize, kernelName ## _kernel);\
  kernelName ## _kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(args);\
}\
void name(const cudaConfig::ExecutionPolicy execPol, ##args){\
  name ## _kernel<<<execPol.getGridSize(), execPol.getBlockSize()>>>(args);\
}\
cudaConfig::ExecutionPolicy name ## _config(bool cudaStatus, int launchSize){\
  return cudaConfig::configure(cudaStatus, launchSize, name ## _kernel);\
}
//@@@@@@@@@@@@@@@@@@@@@@@@@
